#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>

#include "cuda_gemm.hpp"
#include "cuda_gemm_utils.cuh"
#include "cuda_gemm_utils.hpp"

// GEMM kernel v06.
// Each thread in the block processes THREAD_TILE_SIZE_Y *
// THREAD_TILE_SIZE_X output values. Number of threads BLOCK_TILE_SIZE_Y *
// BLOCK_TILE_SIZE_X / (THREAD_TILE_SIZE_Y * THREAD_TILE_SIZE_X)
template <typename T, size_t BLOCK_TILE_SIZE_X, size_t BLOCK_TILE_SIZE_Y,
          size_t BLOCK_TILE_SIZE_K, size_t WARP_TILE_SIZE_X,
          size_t WARP_TILE_SIZE_Y, size_t THREAD_TILE_SIZE_X,
          size_t THREAD_TILE_SIZE_Y, size_t NUM_THREADS_PER_WARP_X,
          size_t NUM_THREADS_PER_WARP_Y>
__global__ void gemm_v08_vectorized(size_t m, size_t n, size_t k, T alpha,
                                    T const* A, size_t lda, T const* B,
                                    size_t ldb, T beta, T* C, size_t ldc)
{
    static_assert(NUM_THREADS_PER_WARP_X * NUM_THREADS_PER_WARP_Y == 32U);
    constexpr size_t NUM_WARPS_X{BLOCK_TILE_SIZE_X / WARP_TILE_SIZE_X};
    static_assert(BLOCK_TILE_SIZE_X % WARP_TILE_SIZE_X == 0U);
    constexpr size_t NUM_WARPS_Y{BLOCK_TILE_SIZE_Y / WARP_TILE_SIZE_Y};
    static_assert(BLOCK_TILE_SIZE_Y % WARP_TILE_SIZE_Y == 0U);
    constexpr unsigned int NUM_THREAD_TILES_PER_WARP_X{
        WARP_TILE_SIZE_X / (THREAD_TILE_SIZE_X * NUM_THREADS_PER_WARP_X)};
    constexpr unsigned int NUM_THREAD_TILES_PER_WARP_Y{
        WARP_TILE_SIZE_Y / (THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y)};
    static_assert(
        WARP_TILE_SIZE_X % (THREAD_TILE_SIZE_X * NUM_THREADS_PER_WARP_X) == 0U);
    static_assert(
        WARP_TILE_SIZE_Y % (THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y) == 0U);

    constexpr unsigned int NUM_THREADS_X{NUM_WARPS_X * NUM_THREADS_PER_WARP_X};
    constexpr unsigned int NUM_THREADS_Y{NUM_WARPS_Y * NUM_THREADS_PER_WARP_Y};
    // Avoid using blockDim.x * blockDim.y as the number of threads per block.
    // Because it is a runtime constant and the compiler cannot optimize the
    // loop unrolling based on that.
    // Use a compile time constant instead.
    constexpr size_t NUM_THREADS{NUM_THREADS_X * NUM_THREADS_Y};

    // Cache a tile of A and B in shared memory for data reuse.
    __shared__ T
        A_thread_block_tile_transposed[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_Y];
    __shared__ T B_thread_block_tile[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_X];

    // A_vals is cached in the register.
    T A_vals[NUM_THREAD_TILES_PER_WARP_Y][THREAD_TILE_SIZE_Y] = {
        static_cast<T>(0)};
    // B_vals is cached in the register.
    T B_vals[NUM_THREAD_TILES_PER_WARP_X][THREAD_TILE_SIZE_X] = {
        static_cast<T>(0)};

    size_t const thread_linear_idx{threadIdx.y * blockDim.x + threadIdx.x};
    size_t const warp_linear_idx{thread_linear_idx / 32U};
    size_t const warp_row_idx{warp_linear_idx / NUM_WARPS_X};
    size_t const warp_col_idx{warp_linear_idx % NUM_WARPS_X};
    size_t const thread_linear_idx_in_warp{thread_linear_idx % 32U};
    size_t const thread_linear_row_idx_in_warp{thread_linear_idx_in_warp /
                                               NUM_THREADS_PER_WARP_X};
    size_t const thread_linear_col_idx_in_warp{thread_linear_idx_in_warp %
                                               NUM_THREADS_PER_WARP_X};

    // Number of outer loops to perform the sum of inner products.
    // C_thread_block_tile =
    // \sigma_{thread_block_tile_idx=0}^{num_thread_block_tiles-1} A[:,
    // thread_block_tile_idx:BLOCK_TILE_SIZE_K] *
    // B[thread_block_tile_idx:BLOCK_TILE_SIZE_K, :]
    size_t const num_thread_block_tiles{(k + BLOCK_TILE_SIZE_K - 1) /
                                        BLOCK_TILE_SIZE_K};
    // Each thread in the block processes NUM_THREAD_TILES_PER_WARP_Y *
    // NUM_THREAD_TILES_PER_WARP_X * THREAD_TILE_SIZE_Y *
    // THREAD_TILE_SIZE_X output values.
    T C_thread_results[NUM_THREAD_TILES_PER_WARP_Y][NUM_THREAD_TILES_PER_WARP_X]
                      [THREAD_TILE_SIZE_Y][THREAD_TILE_SIZE_X] = {
                          static_cast<T>(0)};

    constexpr size_t NUM_VECTOR_UNITS{sizeof(int4) / sizeof(T)};
    static_assert(sizeof(int4) % sizeof(T) == 0U);
    static_assert(BLOCK_TILE_SIZE_K % NUM_VECTOR_UNITS == 0U);
    static_assert(BLOCK_TILE_SIZE_X % NUM_VECTOR_UNITS == 0U);
    constexpr size_t VECTORIZED_THREAD_TILE_SIZE_X{THREAD_TILE_SIZE_X /
                                                   NUM_VECTOR_UNITS};
    static_assert(THREAD_TILE_SIZE_X % NUM_VECTOR_UNITS == 0U);
    constexpr size_t VECTORIZED_THREAD_TILE_SIZE_Y{THREAD_TILE_SIZE_Y /
                                                   NUM_VECTOR_UNITS};
    static_assert(THREAD_TILE_SIZE_Y % NUM_VECTOR_UNITS == 0U);

    for (size_t thread_block_tile_idx{0U};
         thread_block_tile_idx < num_thread_block_tiles;
         ++thread_block_tile_idx)
    {
        load_data_to_shared_memory_transposed_vectorized<
            T, BLOCK_TILE_SIZE_X, BLOCK_TILE_SIZE_Y, BLOCK_TILE_SIZE_K,
            NUM_THREADS>(A, lda, B, ldb, A_thread_block_tile_transposed,
                         B_thread_block_tile, thread_block_tile_idx,
                         thread_linear_idx, m, n, k);
        __syncthreads();

// Perform A[:, thread_block_tile_idx:BLOCK_TILE_SIZE_K] *
// B[thread_block_tile_idx:BLOCK_TILE_SIZE_K, :] where A[:,
// thread_block_tile_idx:BLOCK_TILE_SIZE_K] and
// B[thread_block_tile_idx:BLOCK_TILE_SIZE_K, :] are cached in the
// shared memory as A_thread_block_tile and B_thread_block_tile,
// respectively. This inner product is further decomposed to
// BLOCK_TILE_SIZE_K outer products. A_thread_block_tile *
// B_thread_block_tile = \sigma_{k_i=0}^{BLOCK_TILE_SIZE_K-1}
// A_thread_block_tile[:, k_i] @ B_thread_block_tile[k_i, :] Note that
// both A_thread_block_tile and B_thread_block_tile can be cached in the
// register.
#pragma unroll
        for (size_t k_i{0U}; k_i < BLOCK_TILE_SIZE_K; ++k_i)
        {
#pragma unroll
            for (size_t thread_tile_repeat_row_idx{0U};
                 thread_tile_repeat_row_idx < NUM_THREAD_TILES_PER_WARP_Y;
                 ++thread_tile_repeat_row_idx)
            {
                size_t const A_thread_block_tile_row_idx{
                    warp_row_idx * WARP_TILE_SIZE_Y +
                    thread_tile_repeat_row_idx *
                        (WARP_TILE_SIZE_Y / NUM_THREAD_TILES_PER_WARP_Y) +
                    thread_linear_row_idx_in_warp * THREAD_TILE_SIZE_Y};
                size_t const A_thread_block_tile_col_idx{k_i};
#pragma unroll
                for (size_t thread_tile_y_vector_idx{0U};
                     thread_tile_y_vector_idx < VECTORIZED_THREAD_TILE_SIZE_Y;
                     ++thread_tile_y_vector_idx)
                {
                    *reinterpret_cast<int4*>(
                        &A_vals[thread_tile_repeat_row_idx]
                               [thread_tile_y_vector_idx * NUM_VECTOR_UNITS]) =
                        *reinterpret_cast<int4 const*>(
                            &A_thread_block_tile_transposed
                                [A_thread_block_tile_col_idx]
                                [A_thread_block_tile_row_idx +
                                 thread_tile_y_vector_idx * NUM_VECTOR_UNITS]);
                }
            }
#pragma unroll
            for (size_t thread_tile_repeat_col_idx{0U};
                 thread_tile_repeat_col_idx < NUM_THREAD_TILES_PER_WARP_X;
                 ++thread_tile_repeat_col_idx)
            {
                size_t const B_thread_block_tile_row_idx{k_i};
                size_t const B_thread_block_tile_col_idx{
                    warp_col_idx * WARP_TILE_SIZE_X +
                    thread_tile_repeat_col_idx *
                        (WARP_TILE_SIZE_X / NUM_THREAD_TILES_PER_WARP_X) +
                    thread_linear_col_idx_in_warp * THREAD_TILE_SIZE_X};
#pragma unroll
                for (size_t thread_tile_x_vector_idx{0U};
                     thread_tile_x_vector_idx < VECTORIZED_THREAD_TILE_SIZE_X;
                     ++thread_tile_x_vector_idx)
                {
                    *reinterpret_cast<int4*>(
                        &B_vals[thread_tile_repeat_col_idx]
                               [thread_tile_x_vector_idx * NUM_VECTOR_UNITS]) =
                        *reinterpret_cast<int4 const*>(
                            &B_thread_block_tile[B_thread_block_tile_row_idx]
                                                [B_thread_block_tile_col_idx +
                                                 thread_tile_x_vector_idx *
                                                     NUM_VECTOR_UNITS]);
                }
            }

// Compute NUM_THREAD_TILES_PER_WARP_Y * NUM_THREAD_TILES_PER_WARP_X outer
// products.
#pragma unroll
            for (size_t thread_tile_repeat_row_idx{0U};
                 thread_tile_repeat_row_idx < NUM_THREAD_TILES_PER_WARP_Y;
                 ++thread_tile_repeat_row_idx)
            {
#pragma unroll
                for (size_t thread_tile_repeat_col_idx{0U};
                     thread_tile_repeat_col_idx < NUM_THREAD_TILES_PER_WARP_X;
                     ++thread_tile_repeat_col_idx)
                {
#pragma unroll
                    for (size_t thread_tile_y_idx{0U};
                         thread_tile_y_idx < THREAD_TILE_SIZE_Y;
                         ++thread_tile_y_idx)
                    {
#pragma unroll
                        for (size_t thread_tile_x_idx{0U};
                             thread_tile_x_idx < THREAD_TILE_SIZE_X;
                             ++thread_tile_x_idx)
                        {
                            C_thread_results[thread_tile_repeat_row_idx]
                                            [thread_tile_repeat_col_idx]
                                            [thread_tile_y_idx]
                                            [thread_tile_x_idx] +=
                                A_vals[thread_tile_repeat_row_idx]
                                      [thread_tile_y_idx] *
                                B_vals[thread_tile_repeat_col_idx]
                                      [thread_tile_x_idx];
                        }
                    }
                }
            }
        }
        __syncthreads();
    }

// Write the results to DRAM.
#pragma unroll
    for (size_t thread_tile_repeat_row_idx{0U};
         thread_tile_repeat_row_idx < NUM_THREAD_TILES_PER_WARP_Y;
         ++thread_tile_repeat_row_idx)
    {
#pragma unroll
        for (size_t thread_tile_repeat_col_idx{0U};
             thread_tile_repeat_col_idx < NUM_THREAD_TILES_PER_WARP_X;
             ++thread_tile_repeat_col_idx)
        {
#pragma unroll
            for (size_t thread_tile_y_idx{0U};
                 thread_tile_y_idx < THREAD_TILE_SIZE_Y; ++thread_tile_y_idx)
            {
#pragma unroll
                for (size_t thread_tile_x_vector_idx{0U};
                     thread_tile_x_vector_idx < VECTORIZED_THREAD_TILE_SIZE_X;
                     ++thread_tile_x_vector_idx)
                {
                    size_t const C_row_idx{
                        blockIdx.y * BLOCK_TILE_SIZE_Y +
                        warp_row_idx * WARP_TILE_SIZE_Y +
                        thread_tile_repeat_row_idx *
                            (WARP_TILE_SIZE_Y / NUM_THREAD_TILES_PER_WARP_Y) +
                        thread_linear_row_idx_in_warp * THREAD_TILE_SIZE_Y +
                        thread_tile_y_idx};
                    size_t const C_col_idx{
                        blockIdx.x * BLOCK_TILE_SIZE_X +
                        warp_col_idx * WARP_TILE_SIZE_X +
                        thread_tile_repeat_col_idx *
                            (WARP_TILE_SIZE_X / NUM_THREAD_TILES_PER_WARP_X) +
                        thread_linear_col_idx_in_warp * THREAD_TILE_SIZE_X +
                        thread_tile_x_vector_idx * NUM_VECTOR_UNITS};

                    if (C_row_idx < m && C_col_idx < n)
                    {
                        int4 C_vals{*reinterpret_cast<int4 const*>(
                            &C[C_row_idx * ldc + C_col_idx])};
#pragma unroll
                        for (size_t i{0U}; i < NUM_VECTOR_UNITS; ++i)
                        {
                            reinterpret_cast<T*>(&C_vals)[i] =
                                alpha *
                                    C_thread_results[thread_tile_repeat_row_idx]
                                                    [thread_tile_repeat_col_idx]
                                                    [thread_tile_y_idx]
                                                    [thread_tile_x_vector_idx *
                                                         NUM_VECTOR_UNITS +
                                                     i] +
                                beta * reinterpret_cast<T const*>(&C_vals)[i];
                        }
                        *reinterpret_cast<int4*>(
                            &C[C_row_idx * ldc + C_col_idx]) = C_vals;
                    }
                }
            }
        }
    }
}

template <typename T>
void launch_gemm_kernel_v08_vectorized(size_t m, size_t n, size_t k,
                                       T const* alpha, T const* A, size_t lda,
                                       T const* B, size_t ldb, T const* beta,
                                       T* C, size_t ldc, hipStream_t stream)
{
    // Feel free to play with the block tile sizes.
    // The algorithm correctness should always be guaranteed.
    constexpr unsigned int BLOCK_TILE_SIZE_X{128U};
    constexpr unsigned int BLOCK_TILE_SIZE_Y{128U};
    constexpr unsigned int BLOCK_TILE_SIZE_K{16U};

    constexpr unsigned int WARP_TILE_SIZE_X{32U};
    constexpr unsigned int WARP_TILE_SIZE_Y{64U};
    constexpr unsigned int NUM_WARPS_X{BLOCK_TILE_SIZE_X / WARP_TILE_SIZE_X};
    constexpr unsigned int NUM_WARPS_Y{BLOCK_TILE_SIZE_Y / WARP_TILE_SIZE_Y};
    static_assert(BLOCK_TILE_SIZE_X % WARP_TILE_SIZE_X == 0U);
    static_assert(BLOCK_TILE_SIZE_Y % WARP_TILE_SIZE_Y == 0U);

    constexpr unsigned int THREAD_TILE_SIZE_X{8U};
    constexpr unsigned int THREAD_TILE_SIZE_Y{8U};

    constexpr unsigned int NUM_THREADS_PER_WARP_X{4U};
    constexpr unsigned int NUM_THREADS_PER_WARP_Y{8U};
    static_assert(NUM_THREADS_PER_WARP_X * NUM_THREADS_PER_WARP_Y == 32U);
    static_assert(
        WARP_TILE_SIZE_X % (THREAD_TILE_SIZE_X * NUM_THREADS_PER_WARP_X) == 0U);
    static_assert(
        WARP_TILE_SIZE_Y % (THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y) == 0U);

    constexpr unsigned int NUM_THREADS_X{NUM_WARPS_X * NUM_THREADS_PER_WARP_X};
    constexpr unsigned int NUM_THREADS_Y{NUM_WARPS_Y * NUM_THREADS_PER_WARP_Y};

    constexpr unsigned int NUM_THREADS_PER_BLOCK{NUM_THREADS_X * NUM_THREADS_Y};

    dim3 const block_dim{NUM_THREADS_PER_BLOCK, 1U, 1U};
    dim3 const grid_dim{
        (static_cast<unsigned int>(n) + BLOCK_TILE_SIZE_X - 1U) /
            BLOCK_TILE_SIZE_X,
        (static_cast<unsigned int>(m) + BLOCK_TILE_SIZE_Y - 1U) /
            BLOCK_TILE_SIZE_Y,
        1U};
    gemm_v08_vectorized<T, BLOCK_TILE_SIZE_X, BLOCK_TILE_SIZE_Y,
                        BLOCK_TILE_SIZE_K, WARP_TILE_SIZE_X, WARP_TILE_SIZE_Y,
                        THREAD_TILE_SIZE_X, THREAD_TILE_SIZE_Y,
                        NUM_THREADS_PER_WARP_X, NUM_THREADS_PER_WARP_Y>
        <<<grid_dim, block_dim, 0U, stream>>>(m, n, k, *alpha, A, lda, B, ldb,
                                              *beta, C, ldc);
    CHECK_LAST_CUDA_ERROR();
}

// Explicit instantiation.
template void launch_gemm_kernel_v08_vectorized<float>(
    size_t m, size_t n, size_t k, float const* alpha, float const* A,
    size_t lda, float const* B, size_t ldb, float const* beta, float* C,
    size_t ldc, hipStream_t stream);
template void launch_gemm_kernel_v08_vectorized<double>(
    size_t m, size_t n, size_t k, double const* alpha, double const* A,
    size_t lda, double const* B, size_t ldb, double const* beta, double* C,
    size_t ldc, hipStream_t stream);
template void launch_gemm_kernel_v08_vectorized<__half>(
    size_t m, size_t n, size_t k, __half const* alpha, __half const* A,
    size_t lda, __half const* B, size_t ldb, __half const* beta, __half* C,
    size_t ldc, hipStream_t stream);